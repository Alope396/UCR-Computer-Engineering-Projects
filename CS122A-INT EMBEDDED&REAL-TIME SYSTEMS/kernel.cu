#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void histo_kernel(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins)
{
	
    /*************************************************************************/
    // INSERT KERNEL CODE HERE
	__shared__ unsigned int private_histo[4096]; 
        int j= threadIdx.x; 
        while (j < 4096) { 
         __syncthreads(); 
                private_histo[j] = 0; 
                j += blockDim.x; 
        } 
         __syncthreads();
        int i = threadIdx.x + blockIdx.x * blockDim.x; 
        int stride = blockDim.x * gridDim.x; 
        while (i < size) {    
                atomicAdd( &(private_histo[input[i]]), 1 ); 
                i += stride; 
        } 
        __syncthreads();        
        j = threadIdx.x; 
        while (j < 4096) {    
                atomicAdd(&(histo[j]), private_histo[j] ); 
                j += blockDim.x; 
        } 
        __syncthreads();
	
	  /*************************************************************************/
}

void histogram(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins) {

	  /*************************************************************************/
    //INSERT CODE HERE
        dim3 DimGrid(16, 16, 1);
        dim3 DimBlock(512, 1, 1);

	  /*************************************************************************/

}